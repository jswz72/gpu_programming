#include "hip/hip_runtime.h"
/**
  Jacob Sword
**/
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <cmath>
#include <cassert>
#include <cstdlib>
#include <time.h>
#include "./error_handler.h"
#include "./wtime.h"

using std::cout;
using std::endl;

int sum_cpu(int *arr, int size) {
    int sum = 0;
    for (int i = 0; i < size; i++)
        sum += arr[i];
    return sum;
}

// Every thread atomoically adds its integers to global sum
__global__ void sum_naive_kernel(int *arr, int size, int *sum) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < size) {
        atomicAdd(sum, arr[tid]);
        tid += blockDim.x * gridDim.x;
    }
}

// Every threads gets local sum, smemm used to get block sums.
// Block sums atomically summed to total sum
__global__ void sum_improved_atomic_kernel(int *arr, int size, int *sum) {
    int num_threads = blockDim.x * gridDim.x;
    int division = (size / num_threads) + (size % num_threads != 0);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int start_idx = tid * division;
    int end_idx = (tid + 1) * division;

    // Each thread finds local sum of its assigned area
    int my_sum = 0;
    __shared__ int smem[128];
    for (int i = start_idx; i < end_idx && i < size; i++)
        my_sum += arr[i];
    smem[threadIdx.x] = my_sum;

    // Barrier then use parallel reduction to get block sum
    __syncthreads();
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
            smem[threadIdx.x] = temp;
        }
        __syncthreads();
    }
    // Block sum atomically added to global sum
    if (threadIdx.x == 0) {
        atomicAdd(sum, smem[0]);
    }
}

// Every threads gets local sum, smemm used to get block sums.
__global__ void block_sum_kernel(int *arr, int size, int *block_sums) {
    int num_threads = blockDim.x * gridDim.x;
    int division = (size / num_threads) + (size % num_threads != 0);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int start_idx = tid * division;
    int end_idx = (tid + 1) * division;

    // Each thread finds local sum of its assigned area
    int my_sum = 0;
    __shared__ int smem[128];
    for (int i = start_idx; i < end_idx && i < size; i++)
        my_sum += arr[i];
    smem[threadIdx.x] = my_sum;

    // Barrier then use parallel reduction to get block sum
    __syncthreads();
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
            smem[threadIdx.x] = temp;
        }
        __syncthreads();
    }
    // Block sum added to global arr
    if (threadIdx.x == 0) {
        block_sums[blockIdx.x] = smem[0];
    }
}

int main(int argc, char **argv) {
    int arr_size = 16 * pow(10, 6);
    cout << "Using array size of " << arr_size << endl;
    int *arr = (int *) malloc(sizeof(int) * arr_size);

    srand(time(NULL));

    // Initialize arr
    for (int i = 0; i < arr_size; i++)
        arr[i] = 1 + (rand() % 4);

    int *arr_d;
    HANDLE_ERR(hipMalloc((void **) &arr_d, sizeof (int) * arr_size));
    HANDLE_ERR(hipMemcpy (arr_d, arr, sizeof (int) * arr_size, hipMemcpyHostToDevice));

    int *sum_d;
    HANDLE_ERR(hipMalloc((void **) &sum_d, sizeof (int)));

    // NAIVE GPU
    std::string naive = "Naive GPU";
    double starttime = wtime();
    sum_naive_kernel <<< 128, 128 >>> (arr_d, arr_size, sum_d);
    hipDeviceSynchronize();
    double endtime = wtime();
    double naive_gpu_time = endtime - starttime;
    cout << "Time for " << naive << ": " << naive_gpu_time << endl;

    int sum;
    HANDLE_ERR(hipMemcpy (&sum, sum_d, sizeof (int), hipMemcpyDeviceToHost));

    // CPU
    starttime = wtime();
    int cpu_sum = sum_cpu(arr, arr_size);
    endtime = wtime();
    double cpu_time = endtime - starttime;
    cout << "Time for cpu summation: " << cpu_time << endl;
    
    // Check sum
    assert(sum == cpu_sum);

    // IMPROVED GPU using atomic add
    std::string improved_1 = "Improved GPU using atomic add";
    // Reset device sum
    HANDLE_ERR(hipMemset(sum_d, 0, sizeof(int)));
    starttime = wtime();
    sum_improved_atomic_kernel <<< 128, 128 >>> (arr_d, arr_size, sum_d);
    hipDeviceSynchronize();
    endtime = wtime();
    double improved_gpu_time = endtime - starttime;
    cout << "Time for " << improved_1 << ": " << improved_gpu_time << endl;

    // Check sum
    sum = 0;
    HANDLE_ERR(hipMemcpy (&sum, sum_d, sizeof (int), hipMemcpyDeviceToHost));
    assert(sum == cpu_sum);

    // IMPROVED GPU using CPU add
    std::string improved_2 = "Improved GPU using CPU add";
    // Create block sum
    int *block_sums_d;
    HANDLE_ERR(hipMalloc((void **) &block_sums_d, sizeof (int) * 128));

    starttime = wtime();
    block_sum_kernel <<< 128, 128 >>> (arr_d, arr_size, block_sums_d);
    hipDeviceSynchronize();
    endtime = wtime();
    double improved_gpu_2_time = endtime - starttime;
    cout << "Time for " << improved_2 << ": " << improved_gpu_2_time << endl;

    // Check sum
    int *block_sums = (int *)malloc(sizeof(int) * 128);
    HANDLE_ERR(hipMemcpy (block_sums, block_sums_d, sizeof (int) * 128, hipMemcpyDeviceToHost));
    sum = 0;
    for (int i = 0; i < 128; i++) {
        sum += block_sums[i];
    }
    assert(sum == cpu_sum);

    // Comparisons
    cout << "\n" << endl;
    cout << naive << " is " << (int)(cpu_time / naive_gpu_time) 
        << " times faster than CPU" << endl;
    cout << improved_1 << " is " << (int)(cpu_time / improved_gpu_time) 
        << " times faster than CPU" << endl;
    cout << improved_2 << " is " << (int)(cpu_time / improved_gpu_2_time) 
        << " times faster than CPU" << endl;
    cout << "\n" << endl;
    cout << improved_1 << " is " << naive_gpu_time / improved_gpu_time
        << " times faster than " << naive << endl;
    cout << naive << " is "  << improved_gpu_time / naive_gpu_time
        << " times faster than " << improved_1 << endl;
    cout << improved_2 << " is " << naive_gpu_time / improved_gpu_2_time
        << " times faster than " << naive << endl;
    cout << naive << " is "  << improved_gpu_2_time / naive_gpu_time
        << " times faster than " << improved_2 << endl;
}
