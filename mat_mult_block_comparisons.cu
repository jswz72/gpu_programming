#include "hip/hip_runtime.h"
/**
  Jacob Sword
  Parallelized multiplication of matrix and matrix of random values given fixed matrix dimensions
  Comparisons in speed between regular block by block mult, using padding to combat unalignment,
  and using mtrix transpose to combat strided memory access.
  Matrix dimensions are (256 rows by 240 cols) x (240 rows by 512 cols)
**/
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cassert>
#include <cstdlib>
#include "wtime.h"
#include "./error_handler.h"

using std::cout;
using std::endl;

const int A_ROWS = 256;
const int A_COLS = 240;

const int B_ROWS = 240;
const int B_COLS = 512;

const int C_ROWS = A_ROWS;
const int C_COLS = B_COLS;


//Sequential mat_mult for testing
void mat_mult(int *mat_a, int *mat_b, int *result, int a_rows, int a_cols, int b_cols, int padding = 0)
{
    for (int i = 0; i < a_rows; i++) {
        for (int j = 0; j < b_cols; j++) {
            int temp_res = 0;
            for (int k = 0; k < a_cols; k++) {
                temp_res += mat_a[i * (a_cols + padding) + k] * mat_b[k * b_cols + j];
            }
            result[i * b_cols + j] = temp_res;
        }
    }
}

/*Parallel implementation of matrix a x matrix b
 * 1 block per row
 * matrix A is 256 x 240, matrix b is 240 * 512
 * resultant matrix is 256 rows x 512 cols
 * Supports optional padding of matrix a
 */
__global__ void mat_mult_kernel(int *mat_a, int *mat_b, int *res, 
        int a_rows, int a_cols, int b_cols, int padding = 0) {
    // El for each thread, shared per block
    __shared__ int smem[128];
    for (int row_block = 0; row_block * gridDim.x < a_rows; row_block++) {

        int a_row = blockIdx.x + (row_block * gridDim.x);
        for (int b_col = 0; b_col < b_cols; b_col++) {

            int total = 0;
            for (int thread_i = 0; thread_i * blockDim.x < a_cols; thread_i++) {

                int thread_col = threadIdx.x + (thread_i * blockDim.x);
                // Need to check because 240 not even multiple of 128
                if (thread_col >= a_cols)
                    smem[threadIdx.x] = 0;
                else
                    smem[threadIdx.x] = mat_a[a_row * (a_cols + padding) + thread_col] * mat_b[thread_col * b_cols + b_col];
                __syncthreads();

                //Parallel reduction
                for (int i = blockDim.x / 2; i > 0; i /= 2) {
                    if (threadIdx.x < i) {
                        int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
                        smem[threadIdx.x] = temp;
                    }
                    __syncthreads();
                }
                if (threadIdx.x == 0) {
                    total += smem[threadIdx.x];
                }
            }
            if (threadIdx.x == 0) {
                res[a_row * b_cols + b_col] = total;
            }
        }
    }
}


__global__ void mat_mult_transposed_kernel(int *mat_a, int *mat_b, int *res) {
    int B_TRANS_ROWS = B_COLS;
    int B_TRANS_COLS = B_ROWS;
    // El for each thread, shared per block
    __shared__ int smem[128];
    for (int row_block = 0; row_block * gridDim.x < A_ROWS; row_block++) {

        int a_row = blockIdx.x + (row_block * gridDim.x);
        for (int b_row = 0; b_row < B_TRANS_ROWS; b_row++) {

            int total = 0;
            for (int thread_i = 0; thread_i * blockDim.x < A_COLS; thread_i++) {

                int thread_col = threadIdx.x + (thread_i * blockDim.x);
                // Need to check because 240 not even multiple of 128
                if (thread_col >= A_COLS)
                    smem[threadIdx.x] = 0;
                else
                    smem[threadIdx.x] = mat_a[a_row * A_COLS + thread_col] * mat_b[b_row * B_TRANS_COLS + thread_col];
                __syncthreads();

                //Parallel reduction
                for (int i = blockDim.x / 2; i > 0; i /= 2) {
                    if (threadIdx.x < i) {
                        int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
                        smem[threadIdx.x] = temp;
                    }
                    __syncthreads();
                }
                if (threadIdx.x == 0) {
                    total += smem[threadIdx.x];
                }
            }
            if (threadIdx.x == 0) {
                res[a_row * C_COLS + b_row] = total;
            }
        }
    }
}

void basic_multiplication() {
    int *a = (int *) malloc(sizeof(int) * A_ROWS * A_COLS);
    int *b = (int *) malloc(sizeof(int) * B_ROWS * B_COLS);
    int *c = (int *) malloc(sizeof(int) * C_ROWS * C_COLS);

    srand(time(NULL));
    // Initialize matrix a
    for (int i = 0; i < A_ROWS; i++) {
        for (int j = 0; j < A_COLS; j++) {
            int el = rand() % 10;
            a[i * A_COLS + j] = el;
        }
    }

    // Initialize matrix b
    for (int i = 0; i < B_ROWS; i++) {
        for (int j = 0; j < B_COLS; j++) {
            int el = rand() % 5;
            b[i * B_COLS + j] = el;
        }
    }

    int *a_d, *b_d, *c_d;
    HANDLE_ERR(hipMalloc((void **) &a_d, sizeof (int) * A_ROWS * A_COLS));
    HANDLE_ERR(hipMalloc((void **) &b_d, sizeof (int) * B_ROWS * B_COLS));
    HANDLE_ERR(hipMalloc((void **) &c_d, sizeof (int) * C_ROWS * C_COLS));

    HANDLE_ERR(hipMemcpy (a_d, a, sizeof (int) * A_ROWS * A_COLS, hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy (b_d, b, sizeof (int) * B_ROWS * B_COLS, hipMemcpyHostToDevice));

    double starttime = wtime();
    mat_mult_kernel <<< 128, 128 >>> (a_d, b_d, c_d, A_ROWS, A_COLS, B_COLS);
    hipDeviceSynchronize();
    double algotime = wtime() - starttime;
    cout << "Base multiplication: " << algotime << endl;

    HANDLE_ERR(hipMemcpy (c, c_d, sizeof (int) * C_ROWS * C_COLS, hipMemcpyDeviceToHost));

    //Make sure parallel work is equal to sequential work (for testing)
    int *test_res = (int *) malloc(sizeof(int) * C_ROWS * C_COLS);
    mat_mult(a, b, test_res, A_ROWS, A_COLS, B_COLS);

    for (int i = 0; i < C_ROWS; i++) {
        for (int j = 0; j < C_COLS; j++){
            int idx = i * C_COLS + j;
            if (c[idx] != test_res[idx]) {
                cout << "Not Equal at idx: " << i << ", " << j 
                    << " Parallel work " << c[idx] << ", Sequential Work: " << test_res[idx] << endl;
            }
            assert(c[idx] == test_res[idx]);
        }
    }
}

void padded_multiplication() {
    int padding = 16;
    int A_COLS_PADDED = A_COLS + padding;
    int *a = (int *) malloc(sizeof(int) * A_ROWS * A_COLS_PADDED);
    int *b = (int *) malloc(sizeof(int) * B_ROWS * B_COLS);
    int *c = (int *) malloc(sizeof(int) * C_ROWS * C_COLS);

    srand(time(NULL));
    // Initialize matrix
    for (int i = 0; i < A_ROWS; i++) {
        for (int j = 0; j < A_COLS; j++) {
            int el = rand() % 10;
            a[i * A_COLS_PADDED + j] = el;
        }
    }

    // Initialize vector
    for (int i = 0; i < B_ROWS; i++) {
        for (int j = 0; j < B_COLS; j++) {
            int el = rand() % 5;
            b[i * B_COLS + j] = el;
        }
    }

    int *a_d, *b_d, *c_d;
    HANDLE_ERR(hipMalloc((void **) &a_d, sizeof (int) * A_ROWS * A_COLS_PADDED));
    HANDLE_ERR(hipMalloc((void **) &b_d, sizeof (int) * B_ROWS * B_COLS));
    HANDLE_ERR(hipMalloc((void **) &c_d, sizeof (int) * C_ROWS * C_COLS));

    HANDLE_ERR(hipMemcpy (a_d, a, sizeof (int) * A_ROWS * A_COLS_PADDED, hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy (b_d, b, sizeof (int) * B_ROWS * B_COLS, hipMemcpyHostToDevice));

    double starttime = wtime();
    mat_mult_kernel <<< 128, 128 >>> (a_d, b_d, c_d, A_ROWS, A_COLS, B_COLS, padding);
    hipDeviceSynchronize();
    double algotime = wtime() - starttime;
    cout << "Padded multiplication time: " << algotime << endl;

    HANDLE_ERR(hipMemcpy (c, c_d, sizeof (int) * C_ROWS * C_COLS, hipMemcpyDeviceToHost));

    //Make sure parallel work is equal to sequential work (for testing)
    int *test_res = (int *) malloc(sizeof(int) * C_ROWS * C_COLS);
    mat_mult(a, b, test_res, A_ROWS, A_COLS, B_COLS, padding);

    for (int i = 0; i < C_ROWS; i++) {
        for (int j = 0; j < C_COLS; j++){
            int idx = i * C_COLS + j;
            if (c[idx] != test_res[idx]) {
                cout << "Not Equal at idx: " << i << ", " << j 
                    << " Parallel work " << c[idx] << ", Sequential Work: " << test_res[idx] << endl;
            }
            assert(c[idx] == test_res[idx]);
        }
    }
}

void transpose_multiplication() {
    int *a = (int *) malloc(sizeof(int) * A_ROWS * A_COLS);
    int *b = (int *) malloc(sizeof(int) * B_ROWS * B_COLS);
    int *trans = (int *) malloc(sizeof(int) * B_ROWS * B_COLS);
    int *c = (int *) malloc(sizeof(int) * C_ROWS * C_COLS);

    srand(time(NULL));
    // Initialize matrix a
    for (int i = 0; i < A_ROWS; i++) {
        for (int j = 0; j < A_COLS; j++) {
            int el = rand() % 10;
            a[i * A_COLS + j] = el;
        }
    }

    // Initialize matrix b
    for (int i = 0; i < B_ROWS; i++) {
        for (int j = 0; j < B_COLS; j++) {
            int el = rand() % 5;
            b[i * B_COLS + j] = el;
        }
    }

    // Transpose matrix b
    for (int i = 0; i < B_ROWS; i++) {
        for (int j = 0; j < B_COLS; j++) {
            trans[j * B_ROWS + i] = b[i * B_COLS + j];
        }
    }

    int *a_d, *b_d, *c_d;
    HANDLE_ERR(hipMalloc((void **) &a_d, sizeof (int) * A_ROWS * A_COLS));
    HANDLE_ERR(hipMalloc((void **) &b_d, sizeof (int) * B_ROWS * B_COLS));
    HANDLE_ERR(hipMalloc((void **) &c_d, sizeof (int) * C_ROWS * C_COLS));

    HANDLE_ERR(hipMemcpy (a_d, a, sizeof (int) * A_ROWS * A_COLS, hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy (b_d, trans, sizeof (int) * B_ROWS * B_COLS, hipMemcpyHostToDevice));

    double starttime = wtime();
    mat_mult_transposed_kernel <<< 128, 128 >>> (a_d, b_d, c_d);
    hipDeviceSynchronize();
    double algotime = wtime() - starttime;
    cout << "Transposed multiplication time: " << algotime << endl;

    HANDLE_ERR(hipMemcpy (c, c_d, sizeof (int) * C_ROWS * C_COLS, hipMemcpyDeviceToHost));

    //Make sure parallel work is equal to sequential work (for testing)
    int *test_res = (int *) malloc(sizeof(int) * C_ROWS * C_COLS);
    mat_mult(a, b, test_res, A_ROWS, A_COLS, B_COLS);

    for (int i = 0; i < C_ROWS; i++) {
        for (int j = 0; j < C_COLS; j++){
            int idx = i * C_COLS + j;
            if (c[idx] != test_res[idx]) {
                cout << "Not Equal at idx: " << i << ", " << j 
                    << " Parallel work " << c[idx] << ", Sequential Work: " << test_res[idx] << endl;
            }
            assert(c[idx] == test_res[idx]);
        }
    }
}

int main (int args, char **argv) {
    basic_multiplication();
    padded_multiplication();
    transpose_multiplication();
}
