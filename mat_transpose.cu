#include "hip/hip_runtime.h"
/**
  Jacob Sword
  Parallelized multiplication of matrix and vector of random values given matrix dimensions
**/
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cassert>
#include <cstdlib>
#include <time.h>
#include "./error_handler.h"

using std::cout;
using std::endl;

// Sequential matrix transpose
void mat_transpose(int *mat, int *res, int num_rows, int num_cols)
{
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            res[j * num_rows + i] = mat[i  * num_cols + j];
        }
    }
}

// Parallel matrix tranpose
__global__ void mat_transpose_kernel(int *mat, int *res) {
    int tile_dim = 32;
   __shared__ int smem[32 * 32];

   for (int block_iter = 0; block_iter < 512; block_iter++) {

       // num blocks can fit in "row"
       if (blockIdx.x < 32) {
           local_block_start = blockIdx.x * 32;
       }
       else {
           local_block_start = (tile_dim * tile_dim * 32) + blockIdx.x * 32;
       }
       int block_start = local_block_start + (block_iter * (tile_dim * tile_dim * 64));
       if (threadIdx.x < tile_dim) {
           local_thread_idx = threadIdx.x;
       } else {
           local_thread_idx = ((threadIdx.x / tile_dim) * 32) + (threadIdx.x % tile_dim)

       int block_start = (blockIdx.x * tile_dim);
       if (block_start >= 32) {
           block_start += 
       }
       + (32 * 32 * block_iter);
   }

   int local_tid = threadIdx.x;
   if (local_tid > tile_dim) {
       local_tid += (threadIdx.x / tile_dim) * tile_dim;
       local_tid += threadIdx % tile_dim;
   }

   int block_start = blockIdx.x * tile_dims;
    if (block_start > tile_dim) {
       block_start += blockIdx.x / 
       block_start += threadIdx % tile_dim;
   }
   int idx = blockIdx.x * tile_dims + threadIdx.x;
   if (idx > time_dims) 

}

int main (int args, char **argv) {
    int num_rows = 1024;
    int num_cols = 1024;

    int *a = (int *) malloc(sizeof(int) * num_rows * num_cols);
    int *c = (int *) malloc(sizeof(int) * num_rows * num_cols);

    srand(time(NULL));
    // Initialize matrix
    cout << "Matrix: " << num_rows << " x " << num_cols << endl;
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            int el = rand() % 10;
            a[i * num_cols + j] = el;
        }
    }
    int *a_d, *c_d;
    HANDLE_ERR(hipMalloc((void **) &a_d, sizeof (int) * num_rows * num_cols));
    HANDLE_ERR(hipMalloc((void **) &c_d, sizeof (int) * num_rows * num_cols));

    HANDLE_ERR(hipMemcpy (a_d, a, sizeof (int) * num_rows * num_cols, hipMemcpyHostToDevice));
    mat_transpose_kernel <<< 64, 1024 >>> (a_d, c_d);

    HANDLE_ERR(hipMemcpy (c, c_d, sizeof (int) * num_rows * num_cols, hipMemcpyDeviceToHost));

    //Make sure parallel work is equal to sequential work (for testing)
    int *test_res = (int *) malloc(sizeof(int) * num_rows * num_cols);
    mat_transpose(a, test_res, num_rows, num_cols);

    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++){
            int idx = i * num_cols + j;
            if (c[idx] != test_res[idx]) {
                cout << "Not Equal: " << "Parallel work " << c[idx] 
                    << ", Sequential Work: " << test_res[idx] << endl;
            }
            assert(c[idx] == test_res[idx]);
        }
    }
}
