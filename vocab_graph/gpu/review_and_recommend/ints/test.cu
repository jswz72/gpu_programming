#include "hip/hip_runtime.h"
#include <limits>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <iostream>
#include "graph.h"
#include "error_handler.h"
#include "wtime.h"

using std::cout;
using std::endl;

#define BLOCK_SIZE 16;
#define NUM_ASYNCHRONOUS_ITERATIONS 20  // Number of async loop iterations before attempting to read results back

/***************************/
/* MASKARRAYEMPTY FUNCTION */
/***************************/
// --- Check whether all the vertices have been finalized. This tells the algorithm whether it needs to continue running or not.
bool allFinalizedVertices(bool *finalizedVertices, int numVertices) {

    for (int i = 0; i < numVertices; i++)  if (finalizedVertices[i] == true) { return false; }

    return true;
}

//Round a / b to nearest higher integer value
int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

/*************************/
/* ARRAY INITIALIZATIONS */
/*************************/
__global__ void initializeArrays(bool * __restrict__ d_finalizedVertices, int* __restrict__ d_shortestDistances, int* __restrict__ d_updatingShortestDistances,
                                 const int sourceVertex, const int numVertices) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (sourceVertex == tid) {

            d_finalizedVertices[tid]            = true;
            d_shortestDistances[tid]            = 0;
            d_updatingShortestDistances[tid]    = 0; }

        else {

            d_finalizedVertices[tid]            = false;
            d_shortestDistances[tid]            = INT_MAX;
            d_updatingShortestDistances[tid]    = INT_MAX;
        }
    }
}

/**************************/
/* DIJKSTRA GPU KERNEL #1 */
/**************************/
__global__  void Kernel1(const int * __restrict__ vertexArray, const int* __restrict__ edgeArray,
                         const int * __restrict__ weightArray, bool * __restrict__ finalizedVertices, int* __restrict__ shortestDistances,
                         int * __restrict__ updatingShortestDistances, const int numVertices, const int numEdges) {

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (finalizedVertices[tid] == true) {

            finalizedVertices[tid] = false;

            int edgeStart = vertexArray[tid];
            int edgeEnd = vertexArray[tid + 1];

            for (int edge = edgeStart; edge < edgeEnd; edge++) {
                int nid = edgeArray[edge];
                atomicMin(&updatingShortestDistances[nid], shortestDistances[tid] + weightArray[edge]);
            }
        }
    }
}

/**************************/
/* DIJKSTRA GPU KERNEL #1 */
/**************************/
__global__  void Kernel2(const int * __restrict__ vertexArray, const int * __restrict__ edgeArray, const int* __restrict__ weightArray,
                         bool * __restrict__ finalizedVertices, int* __restrict__ shortestDistances, int* __restrict__ updatingShortestDistances,
                         const int numVertices) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (shortestDistances[tid] > updatingShortestDistances[tid]) {
            shortestDistances[tid] = updatingShortestDistances[tid];
            finalizedVertices[tid] = true; }

        updatingShortestDistances[tid] = shortestDistances[tid];
    }
}

/* dijkstraGPU FUNCTION */
/************************/
void dijkstraGPU(int *beg_pos, int *adj_list, int *weights, const int sourceVertex, int * __restrict__ h_shortestDistances, int num_vtx, int num_edge) {

    // --- Create device-side adjacency-list, namely, vertex array Va, edge array Ea and weight array Wa from G(V,E,W)
    int     *d_beg_pos;         HANDLE_ERR(hipMalloc(&d_beg_pos,    sizeof(int)   *  num_vtx));
    int     *d_adj_list;           HANDLE_ERR(hipMalloc(&d_adj_list,  sizeof(int)   * num_edge));
    int   *d_weights;         HANDLE_ERR(hipMalloc(&d_weights,    sizeof(int) * num_edge));

    // --- Copy adjacency-list to the device
    HANDLE_ERR(hipMemcpy(d_beg_pos, beg_pos, sizeof(int)   * num_vtx, hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy(d_adj_list,   adj_list,   sizeof(int)   * num_edge,    hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy(d_weights, weights, sizeof(int) * num_edge,    hipMemcpyHostToDevice));

    // --- Create mask array Ma, cost array Ca and updating cost array Ua of size V
    bool    *d_finalizedVertices;           HANDLE_ERR(hipMalloc(&d_finalizedVertices,       sizeof(bool)   * num_vtx));
    int   *d_shortestDistances;           HANDLE_ERR(hipMalloc(&d_shortestDistances,       sizeof(int) * num_vtx));
    int   *d_updatingShortestDistances;   HANDLE_ERR(hipMalloc(&d_updatingShortestDistances, sizeof(int) * num_vtx));

    bool *h_finalizedVertices = (bool *)malloc(sizeof(bool) * num_vtx);

    // --- Initialize mask Ma to false, cost array Ca and Updating cost array Ua to \u221e
    initializeArrays 
        <<<iDivUp(num_vtx, 16), 16 >>>
        (d_finalizedVertices, d_shortestDistances, d_updatingShortestDistances, sourceVertex, num_vtx);
    HANDLE_ERR(hipPeekAtLastError());
    HANDLE_ERR(hipDeviceSynchronize());

    // --- Read mask array from device -> host
    HANDLE_ERR(hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * num_vtx, hipMemcpyDeviceToHost));

    while (!allFinalizedVertices(h_finalizedVertices, num_vtx)) {

        // --- In order to improve performance, we run some number of iterations without reading the results.  This might result
        //     in running more iterations than necessary at times, but it will in most cases be faster because we are doing less
        //     stalling of the GPU waiting for results.
        for (int asyncIter = 0; asyncIter < NUM_ASYNCHRONOUS_ITERATIONS; asyncIter++) {

            Kernel1 <<<iDivUp(num_vtx, 16), 16 >>>(d_beg_pos, d_adj_list, d_weights, d_finalizedVertices, d_shortestDistances,
                                                            d_updatingShortestDistances, num_vtx, num_edge);
            HANDLE_ERR(hipPeekAtLastError());
            HANDLE_ERR(hipDeviceSynchronize());
            Kernel2 <<<iDivUp(num_vtx, 16), 16 >>>(d_beg_pos, d_adj_list, d_weights, d_finalizedVertices, d_shortestDistances, d_updatingShortestDistances,
                                                            num_vtx);
            HANDLE_ERR(hipPeekAtLastError());
            HANDLE_ERR(hipDeviceSynchronize());
        }

        HANDLE_ERR(hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * num_vtx, hipMemcpyDeviceToHost));
    }

    // --- Copy the result to host
    HANDLE_ERR(hipMemcpy(h_shortestDistances, d_shortestDistances, sizeof(int) * num_vtx, hipMemcpyDeviceToHost));

    free(h_finalizedVertices);

    HANDLE_ERR(hipFree(d_beg_pos));
    HANDLE_ERR(hipFree(d_adj_list));
    HANDLE_ERR(hipFree(d_weights));
    HANDLE_ERR(hipFree(d_finalizedVertices));
    HANDLE_ERR(hipFree(d_shortestDistances));
    HANDLE_ERR(hipFree(d_updatingShortestDistances));
}

/* minDistance FUNCTION */
/************************/
// --- Finds the vertex with minimum distance value, from the set of vertices not yet included in shortest path tree
int minDistance(int *shortestDistances, bool *finalizedVertices, const int sourceVertex, const int N) {

    // --- Initialize minimum value
    int minIndex = sourceVertex;
    int min = INT_MAX;

    for (int v = 0; v < N; v++)
        if (finalizedVertices[v] == false && shortestDistances[v] <= min) min = shortestDistances[v], minIndex = v;

    return minIndex;
}

void dijkstraCPU(int *beg_pos, int *adj_list, int *weights, int *h_shortestDistances, int sourceVertex, const int N) {

    // --- h_finalizedVertices[i] is true if vertex i is included in the shortest path tree
    //     or the shortest distance from the source node to i is finalized
    bool *h_finalizedVertices = (bool *)malloc(N * sizeof(bool));

    // --- Initialize h_shortestDistancesances as infinite and h_shortestDistances as false
    for (int i = 0; i < N; i++) h_shortestDistances[i] = INT_MAX, h_finalizedVertices[i] = false;

    // --- h_shortestDistancesance of the source vertex from itself is always 0
    h_shortestDistances[sourceVertex] = 0;

    // --- Dijkstra iterations
    for (int iterCount = 0; iterCount < N - 1; iterCount++) {

        // --- Selecting the minimum distance vertex from the set of vertices not yet
        //     processed. currentVertex is always equal to sourceVertex in the first iteration.
        int currentVertex = minDistance(h_shortestDistances, h_finalizedVertices, sourceVertex, N);

        // --- Mark the current vertex as processed
        h_finalizedVertices[currentVertex] = true;

        // --- Relaxation loop
        for (int v = 0; v < N; v++) {

            // --- Update dist[v] only if it is not in h_finalizedVertices, there is an edge
            //     from u to v, and the cost of the path from the source vertex to v through
            //     currentVertex is smaller than the current value of h_shortestDistances[v]
            bool found = false;
            int idx = 0;
            for (int i = beg_pos[currentVertex]; i < beg_pos[currentVertex + 1]; i++) {
                if (v == adj_list[i]) {
                    found = true;
                    idx = i;
                }
            }
            if (!h_finalizedVertices[v] &&
                found &&
                h_shortestDistances[currentVertex] != INT_MAX &&
                h_shortestDistances[currentVertex] + weights[idx] < h_shortestDistances[v]) {

                h_shortestDistances[v] = h_shortestDistances[currentVertex] + weights[idx];
                //cout << weights[idx] << endl;
            }
        }
    }
}

int main(int argc, char **argv) {
    int source_vertex = atoi(argv[1]);
    graph<long, long, double, long, long, double> *csr = 
        new graph<long, long, double, long, long, double>
        ("../../get_edges/edge-list.txt_beg_pos.bin",
         "../../get_edges/edge-list.txt_csr.bin",
         "../../get_edges/edge-list.txt_weight.bin");

    int *shortest_dist_cpu = (int *)malloc(csr->vert_count * sizeof(int));

    int *beg_pos = (int *)malloc(csr->vert_count * sizeof(int));
    int *adj_list = (int *)malloc(csr->edge_count * sizeof(int));
    int *weight = (int *)malloc(csr->edge_count * sizeof(int));

    for (int i = 0; i < csr->vert_count; i++) beg_pos[i] = (int) csr->beg_pos[i];
    
    for (int i = 0; i < csr->edge_count; i++) adj_list[i] = (int) csr->csr[i];
    for (int i = 0; i < csr->edge_count; i++) weight[i] = (int) (csr->weight[i] * 1000);
    for (int i = 0; i < 500; i++) {
        cout << "ASDF: " << weight[i] << endl;
    }

    double cpu_start = wtime();
    cout << "INTMAX: " << INT_MAX << endl;
    dijkstraCPU(beg_pos, adj_list, weight, shortest_dist_cpu, source_vertex, csr->vert_count);
    for (int i = 0; i < csr->vert_count; i++) {
        //cout << shortest_dist_cpu[i] << endl;
    }
    double cpu_end = wtime();
    cout << "CPU completed in " << cpu_end - cpu_start << endl;
    
    int *shortest_dist_gpu = (int *) malloc(csr->vert_count * sizeof(int));
    double gpu_start = wtime();
    dijkstraGPU(beg_pos, adj_list, weight, source_vertex, shortest_dist_gpu, csr->vert_count, csr->edge_count);
    double gpu_end = wtime();
    cout << "GPU completed in " << gpu_end - gpu_start << endl;

    for (int i = 0; i < csr->vert_count; i++) {
        if (shortest_dist_gpu[i] != shortest_dist_cpu[i]) {
            cout << "Index " << i << " fail: " << "CPU " << shortest_dist_cpu[i] 
            << " GPU " << shortest_dist_gpu[i] << endl;
        }
        assert(shortest_dist_gpu[i] == shortest_dist_cpu[i]);
    }
}
