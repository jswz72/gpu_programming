#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <algorithm>
#include <string>
#include <vector>
#include <fstream>
#include <cstdlib>
#include "graph.h"
#include "error_handler.h"
#include "wtime.h"

using std::cout;
using std::endl;
using std::string;

struct WordDist {
    int dist;
    int word_id;
    WordDist(double dist, int id): dist(dist), word_id(id) {};
};

typedef graph<long, long, double, long, long, double> CSR;

std::vector<string> get_word_mapping(const char *mapping_file) {
	std::ifstream infile(mapping_file);
	std::vector<string> words;
	string line;
	while (std::getline(infile, line))
		words.push_back(line);
	return words;
}

__device__ double DOUBLE_MAX = std::numeric_limits<double>::max();
__device__ double DOUBLE_INF = std::numeric_limits<double>::infinity();

// Inverse sum rule, closness of vtx to all sources
__device__ double get_collective_dist(int *dist, int rows, int cols, int col) {
    double sum = 0;
    for (int i = 0; i < rows; i++) {
        sum += 1 / (double)dist[i * cols + col];
    }
    return sum;
}

__device__ long min_dist(int *distances, unsigned int *path, int vert_count)
{
    int min = INT_MAX;
    long min_idx;
    for (int i = 0; i < vert_count; i++)
    {
        if (!path[i] && distances[i] <= min)
        {
            min = distances[i];
            min_idx = i;
        }
    }
    return min_idx;
}

/**
 * Find shortest weighted path to all nodes from source using djikstra's algorithm
 */
__global__ void shortest_path_weights_kernel(long *beg_pos, long *adj_list, int *weight, int vert_count, int source, int *distances)
{
    // bitset true if included in path
    unsigned int *path = new unsigned int[vert_count];
    for (int i = 0; i < vert_count; i++)
    {
        distances[i] = INT_MAX;
        path[i] = 0;
    }

    distances[source] = 0;
    for (int count = 0; count < vert_count - 1; count++)
    {
        long cur = min_dist(distances, path, vert_count);
        path[cur] = true;

        // Update distances
        for (int i = beg_pos[cur]; i < beg_pos[cur+1]; i++)
        {
			int neighbor = adj_list[i];
            if (!path[neighbor] && 
                    distances[cur] != INT_MAX &&
                     distances[cur] + weight[i] < distances[neighbor])
            {
                distances[neighbor] = distances[cur] + weight[i];
            }
        }
    }
}


__global__ void collective_closest_kernel(int *dist, int num_source_words, int vert_count, int *word_ids, int *dists) {

    // Word has no relation to given set
    double no_relation = (1 / (double)INT_MAX) * num_source_words;

    // Get collective dist of vtx (col) to all source words (row)
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = tidx;
    if (!tidx)
        printf("Starting collective closest\n");

    while (tid < vert_count) {
        double my_dist = get_collective_dist(dist, num_source_words, vert_count, tid);
        bool append = my_dist != no_relation && my_dist != DOUBLE_INF;
        if (append) {
            word_ids[tid] = tid;
            dists[tid] = my_dist;
        }
        else {
            word_ids[tid] = -1;
            dists[tid] = -1;
        }
        tid += blockDim.x * gridDim.x;
    }
    if (!tidx)
        printf("Done with collective closest\n");
}

int main(int argc, char **argv) {
	if (argc < 5) {
		cout << "Input: ./exe base_file mapping_file num_recs source_words..." << endl;
		return 1;
	}
	
    // Get files and make graph
	string base_filename (argv[1]);
	string beg_file = base_filename + "_beg_pos.bin";
	string csr_file = base_filename + "_csr.bin";
	string weight_file = base_filename + "_weight.bin";
	const char *mapping_file = argv[2];
	graph<long, long, double, long, long, double> *csr = 
		new graph <long, long, double, long, long, double>
		(beg_file.c_str(), csr_file.c_str(), weight_file.c_str());

    // Output Graph Info
	std::cout << "Edges: " << csr->edge_count << std::endl;
    std::cout << "Verticies: " << csr->vert_count << std::endl;

	int num_recs = atoi(argv[3]);
	int num_source_words = argc - 4;

    // Get source word indices and make sure they are in graph
	std::vector<int> source_word_idxs;
	std::vector<string> words = get_word_mapping(mapping_file);
    for (int i = 0; i < num_source_words; i++) {
		const char *source_word = argv[i + 4];
		auto it = std::find(words.begin(), words.end(), source_word);
		if (it == words.end()) {
			cout << "Not found in graph: " << source_word << endl;
			return 1;
		}
		int idx = std::distance(words.begin(), it);
		source_word_idxs.push_back(idx);
    }

    // Allocate and copy device variables

    // IDXs of source_words
    int *source_idxs_d;
    HANDLE_ERR(hipMalloc((void **) &source_idxs_d, sizeof(int) * source_word_idxs.size()));
    HANDLE_ERR(hipMemcpy (source_idxs_d, source_word_idxs.data(), sizeof(int) * source_word_idxs.size(), hipMemcpyHostToDevice));

    // Beg pos array
    long *beg_pos_d;
    HANDLE_ERR(hipMalloc((void **) &beg_pos_d, sizeof(long) * (csr->vert_count + 1)));
    HANDLE_ERR(hipMemcpy (beg_pos_d, csr->beg_pos, sizeof(long) * (csr->vert_count + 1), hipMemcpyHostToDevice));

    // Adj list
    long *csr_d;
    HANDLE_ERR(hipMalloc((void **) &csr_d, sizeof(long) * csr->edge_count));
    HANDLE_ERR(hipMemcpy (csr_d, csr->csr, sizeof(long) * csr->edge_count, hipMemcpyHostToDevice));

    int *weight_int = (int *)malloc(sizeof(int) * csr->edge_count);
    for (int i = 0; i < csr->edge_count; i++) {
        weight_int[i] = (int) (csr->weight[i] * 1000);
    }

    // Weights array
    int *weight_d;
    HANDLE_ERR(hipMalloc((void **) &weight_d, sizeof(int) * csr->edge_count));
    HANDLE_ERR(hipMemcpy (weight_d, weight_int, sizeof(int) * csr->edge_count, hipMemcpyHostToDevice));

    // Matrix of dists results of sssp
    int *dist_mat_d;
    HANDLE_ERR(hipMalloc((void **) &dist_mat_d, sizeof(int) * csr->vert_count * num_source_words));

    // Word ids of summed array
    int *word_ids_d;
    HANDLE_ERR(hipMalloc((void **) &word_ids_d, sizeof(int) * csr->vert_count));

    // Word dists of summed array
    int *dists_d;
    HANDLE_ERR(hipMalloc((void **) &dists_d, sizeof(int) * csr->vert_count));

    // Number of recommendations to make/made
    int *num_recs_d;
    HANDLE_ERR(hipMalloc((void **) &num_recs_d, sizeof(int)));
    HANDLE_ERR(hipMemcpy (num_recs_d, &num_recs, sizeof(int), hipMemcpyHostToDevice));





    // Just doing it for first word this time
    int tid = 0;
    double starttime = wtime();
    printf("Starting SSSP\n");

    // Repurposing dist_mat_d
    shortest_path_weights_kernel<<<1, 1>>>(beg_pos_d, csr_d, weight_d, csr->vert_count, source_word_idxs[tid], dist_mat_d);
    hipDeviceSynchronize();
    printf("SSSP Done\n");
    double endtime = wtime();
    cout << "Time for SSSP: " << endtime - starttime << endl;

    starttime = wtime();
    collective_closest_kernel <<< 128, 128 >>> (dist_mat_d, num_source_words, csr->vert_count, word_ids_d, dists_d);
    hipDeviceSynchronize();
    endtime = wtime();
    cout << "Time for collective closest: " << endtime - starttime << endl;


    
    
    
    // BELOW THIS SAME 
    // Copy back closest_words
    int *ids = (int *)malloc(sizeof(int*) * csr->vert_count);
    int *dists = (int *)malloc(sizeof(int*) * csr->vert_count);
    HANDLE_ERR(hipMemcpy (dists, dists_d, sizeof(int) * csr->vert_count, hipMemcpyDeviceToHost));
    HANDLE_ERR(hipMemcpy (ids, word_ids_d, sizeof(int) * csr->vert_count, hipMemcpyDeviceToHost));

    WordDist **wd = (WordDist**)malloc(sizeof(WordDist*) * csr->vert_count);
    for (int i = 0; i < csr->vert_count; i++) {
        wd[i] = new WordDist(dists[i], ids[i]);
    }

    // Sort in terms of collect closest
	std::sort(wd, wd + csr->vert_count, [](WordDist *a, WordDist *b) -> bool
    {
        return a->dist > b->dist;
    });

	cout << "\nLearning recommendations :" << endl;
	for (int i = 0; i < num_recs; i++) {
        if (wd[i]->word_id == -1) {
            cout << "End" << endl;
            break;
        }
		cout << words[wd[i]->word_id] << " (Value: "
			<< wd[i]->dist << ")" << endl;
	}
	return 0;	
}
