#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <algorithm>
#include <string>
#include <vector>
#include <fstream>
#include <cstdlib>
#include "graph.h"
#include "error_handler.h"
#include "wtime.h"

using std::cout;
using std::endl;
using std::string;

struct WordDist {
    double dist;
    int word_id;
    WordDist(double dist, int id): dist(dist), word_id(id) {};
};

typedef graph<long, long, double, long, long, double> CSR;

std::vector<string> get_word_mapping(const char *mapping_file) {
	std::ifstream infile(mapping_file);
	std::vector<string> words;
	string line;
	while (std::getline(infile, line))
		words.push_back(line);
	return words;
}

__device__ double DOUBLE_MAX = std::numeric_limits<double>::max();
__device__ double DOUBLE_INF = std::numeric_limits<double>::infinity();

// Inverse sum rule, closness of vtx to all sources
__device__ double get_collective_dist(double *dist, int rows, int cols, int col) {
    double sum = 0;
    for (int i = 0; i < rows; i++) {
        sum += (1 / dist[i * cols + col]);
    }
    return sum;
}

__device__ long min_dist(double *distances, unsigned int *path, int vert_count)
{
    double min = DOUBLE_MAX;
    long min_idx;
    for (int i = 0; i < vert_count; i++)
    {
        if (!path[i] && distances[i] <= min)
        {
            min = distances[i];
            min_idx = i;
        }
    }
    return min_idx;
}

/**
 * Find shortest weighted path to all nodes from source using djikstra's algorithm
 */
__global__ void shortest_path_weights_kernel(long *beg_pos, long *adj_list, double *weight, int vert_count, int source, double *distances)
{
    // bitset true if included in path
    unsigned int *path = new unsigned int[vert_count];
    for (int i = 0; i < vert_count; i++)
    {
        distances[i] = DOUBLE_MAX;
        path[i] = 0;
    }

    distances[source] = 0;
    for (int count = 0; count < vert_count - 1; count++)
    {
        long cur = min_dist(distances, path, vert_count);
        path[cur] = true;

        // Update distances
        for (int i = beg_pos[cur]; i < beg_pos[cur+1]; i++)
        {
			int neighbor = adj_list[i];
            if (!path[neighbor] && 
                    distances[cur] != DOUBLE_MAX &&
                     distances[cur] + weight[i] < distances[neighbor])
            {
                double to_write = distances[cur] + weight[i];
                distances[neighbor] = to_write;
            }
        }
    }
}


__global__ void collective_closest_kernel(double *dist, int num_source_words, int vert_count, int *word_ids, double *dists) {

    // Word has no relation to given set
    double no_relation = (1 / DOUBLE_MAX) * num_source_words;

    // Get collective dist of vtx (col) to all source words (row)
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (!tid)
        printf("Starting collective closest\n");

    while (tid < vert_count) {
        double my_dist = get_collective_dist(dist, num_source_words, vert_count, tid);
        //printf("%d\n", dist[tid]);
        bool append = my_dist != DOUBLE_INF && my_dist != no_relation;
        if (append) {
            word_ids[tid] = tid;
            dists[tid] = my_dist;
        }
        else {
            word_ids[tid] = -1;
            dists[tid] = -1;
        }
        tid += blockDim.x * gridDim.x;
    }
    if (!tid)
        printf("Done with collective closest\n");
}

int main(int argc, char **argv) {
	if (argc < 5) {
		cout << "Input: ./exe base_file mapping_file num_recs source_words..." << endl;
		return 1;
	}
	
    // Get files and make graph
	string base_filename (argv[1]);
	string beg_file = base_filename + "_beg_pos.bin";
	string csr_file = base_filename + "_csr.bin";
	string weight_file = base_filename + "_weight.bin";
	const char *mapping_file = argv[2];
	graph<long, long, double, long, long, double> *csr = 
		new graph <long, long, double, long, long, double>
		(beg_file.c_str(), csr_file.c_str(), weight_file.c_str());

    // Output Graph Info
	std::cout << "Edges: " << csr->edge_count << std::endl;
    std::cout << "Verticies: " << csr->vert_count << std::endl;

	int num_recs = atoi(argv[3]);
	int num_source_words = argc - 4;

    // Get source word indices and make sure they are in graph
	std::vector<int> source_word_idxs;
	std::vector<string> words = get_word_mapping(mapping_file);
    for (int i = 0; i < num_source_words; i++) {
		const char *source_word = argv[i + 4];
		auto it = std::find(words.begin(), words.end(), source_word);
		if (it == words.end()) {
			cout << "Not found in graph: " << source_word << endl;
			return 1;
		}
		int idx = std::distance(words.begin(), it);
		source_word_idxs.push_back(idx);
    }

    // Allocate and copy device variables

    // IDXs of source_words
    int *source_idxs_d;
    HANDLE_ERR(hipMalloc((void **) &source_idxs_d, sizeof(int) * source_word_idxs.size()));
    HANDLE_ERR(hipMemcpy (source_idxs_d, source_word_idxs.data(), sizeof(int) * source_word_idxs.size(), hipMemcpyHostToDevice));

    // Beg pos array
    long *beg_pos_d;
    HANDLE_ERR(hipMalloc((void **) &beg_pos_d, sizeof(long) * (csr->vert_count + 1)));
    HANDLE_ERR(hipMemcpy (beg_pos_d, csr->beg_pos, sizeof(long) * (csr->vert_count + 1), hipMemcpyHostToDevice));

    // Adj list
    long *csr_d;
    HANDLE_ERR(hipMalloc((void **) &csr_d, sizeof(long) * csr->edge_count));
    HANDLE_ERR(hipMemcpy (csr_d, csr->csr, sizeof(long) * csr->edge_count, hipMemcpyHostToDevice));

    // Weights array
    double *weight_d;
    HANDLE_ERR(hipMalloc((void **) &weight_d, sizeof(double) * csr->edge_count));
    HANDLE_ERR(hipMemcpy (weight_d, csr->weight, sizeof(double) * csr->edge_count, hipMemcpyHostToDevice));

    // Matrix of dists results of sssp
    double *dist_mat_d;
    HANDLE_ERR(hipMalloc((void **) &dist_mat_d, sizeof(double) * csr->vert_count * num_source_words));

    // Word ids of summed array
    int *word_ids_d;
    HANDLE_ERR(hipMalloc((void **) &word_ids_d, sizeof(int) * csr->vert_count));

    // Word dists of summed array
    double *dists_d;
    HANDLE_ERR(hipMalloc((void **) &dists_d, sizeof(double) * csr->vert_count));

    // Number of recommendations to make/made
    int *num_recs_d;
    HANDLE_ERR(hipMalloc((void **) &num_recs_d, sizeof(int)));
    HANDLE_ERR(hipMemcpy (num_recs_d, &num_recs, sizeof(int), hipMemcpyHostToDevice));





    // Just doing it for first word this time
    int tid = 0;
    double starttime = wtime();
    printf("Starting SSSP\n");

    // Repurposing dist_mat_d
    printf("num sw: %d\n", num_source_words);
    printf("asdf: %d\n", source_word_idxs[tid]);
    shortest_path_weights_kernel<<<1, 1>>>(beg_pos_d, csr_d, weight_d, csr->vert_count, source_word_idxs[tid], dist_mat_d);
    hipDeviceSynchronize();
    printf("SSSP Done\n");
    double endtime = wtime();
    cout << "Time for SSSP: " << endtime - starttime << endl;

    collective_closest_kernel <<< 128, 128 >>> (dist_mat_d, num_source_words, csr->vert_count, word_ids_d, dists_d);
    hipDeviceSynchronize();


    
    
    
    // BELOW THIS SAME 
    // Copy back closest_words
    int *ids = (int *)malloc(sizeof(int*) * csr->vert_count);
    double *dists = (double *)malloc(sizeof(double*) * csr->vert_count);
    HANDLE_ERR(hipMemcpy (dists, dists_d, sizeof(double) * csr->vert_count, hipMemcpyDeviceToHost));
    HANDLE_ERR(hipMemcpy (ids, word_ids_d, sizeof(int) * csr->vert_count, hipMemcpyDeviceToHost));

    WordDist **wd = (WordDist**)malloc(sizeof(WordDist*) * csr->vert_count);
    for (int i = 0; i < csr->vert_count; i++) {
        wd[i] = new WordDist(dists[i], ids[i]);
    }

    // Sort in terms of collect closest
	std::sort(wd, wd + csr->vert_count, [](WordDist *a, WordDist *b) -> bool
    {
        return a->dist > b->dist;
    });

	cout << "\nLearning recommendations :" << endl;
	for (int i = 0; i < num_recs; i++) {
        if (wd[i]->word_id == -1) {
            cout << "End" << endl;
            break;
        }
		cout << words[wd[i]->word_id] << " (Value: "
			<< wd[i]->dist << ")" << endl;
	}
	return 0;	
}
